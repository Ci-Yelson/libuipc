#include <affine_body/affine_body_constraint.h>
#include <affine_body/utils.h>
#include <uipc/builtin/attribute_name.h>
#include <kernel_cout.h>
#include <animator/utils.h>

namespace uipc::backend::cuda
{
class SoftTransformConstraint final : public AffineBodyConstraint
{
    static constexpr U64 SoftTransformConstraintUID = 16ull;

  public:
    using AffineBodyConstraint::AffineBodyConstraint;

    vector<IndexT>   h_constrained_bodies;
    vector<Vector12> h_aim_transforms;
    vector<Vector2>  h_strength_ratios;

    muda::DeviceBuffer<IndexT>   constrained_bodies;
    muda::DeviceBuffer<Vector12> aim_transforms;
    muda::DeviceBuffer<Vector2>  strength_ratios;

    virtual void do_build(BuildInfo& info) override {}

    virtual U64 get_uid() const noexcept override
    {
        return SoftTransformConstraintUID;
    }

    void do_init(AffineBodyAnimator::FilteredInfo& info) override
    {
        auto count = info.anim_body_count();

        // reserve memory
        h_constrained_bodies.reserve(count);
        h_aim_transforms.reserve(count);
        h_strength_ratios.reserve(count);
        constrained_bodies.reserve(count);
        aim_transforms.reserve(count);
        strength_ratios.reserve(count);

        do_step(info);  // do the same thing as do_step
    }

    void do_step(AffineBodyAnimator::FilteredInfo& info) override
    {
        using ForEachInfo = AffineBodyDynamics::ForEachInfo;

        auto geo_slots = world().scene().geometries();

        // clear
        h_constrained_bodies.clear();
        h_aim_transforms.clear();
        h_strength_ratios.clear();

        IndexT current_body_offset = 0;
        info.for_each(
            geo_slots,
            [&](geometry::SimplicialComplex& sc)
            {
                auto body_offset = sc.meta().find<IndexT>(builtin::backend_abd_body_offset);
                current_body_offset = body_offset->view().front();

                auto is_constrained = sc.instances().find<IndexT>(builtin::is_constrained);
                auto aim_transform = sc.instances().find<Matrix4x4>(builtin::aim_transform);
                auto strength_ratio = sc.instances().find<Vector2>("strength_ratio");

                return zip(is_constrained->view(),
                           aim_transform->view(),
                           strength_ratio->view());
            },
            [&](const ForEachInfo& I, auto&& values)
            {
                SizeT bI = I.local_index() + current_body_offset;

                auto&& [is_constrained, aim_transform, strength_ratio] = values;

                if(is_constrained)
                {
                    h_constrained_bodies.push_back(bI);
                    Vector12 q = transform_to_q(aim_transform);
                    h_aim_transforms.push_back(q);
                    h_strength_ratios.push_back(strength_ratio);
                }
            });

        constrained_bodies.resize(h_constrained_bodies.size());
        constrained_bodies.view().copy_from(h_constrained_bodies.data());

        aim_transforms.resize(h_aim_transforms.size());
        aim_transforms.view().copy_from(h_aim_transforms.data());

        strength_ratios.resize(h_strength_ratios.size());
        strength_ratios.view().copy_from(h_strength_ratios.data());
    }

    void do_report_extent(AffineBodyAnimator::ReportExtentInfo& info) override
    {
        info.energy_count(h_constrained_bodies.size());
        info.gradient_segment_count(h_constrained_bodies.size());
        info.hessian_block_count(h_constrained_bodies.size());
    }

    void do_compute_energy(AffineBodyAnimator::EnergyInfo& info) override
    {
        using namespace muda;

        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(constrained_bodies.size(),
                   [substep_ratio = info.substep_ratio(),
                    indices       = constrained_bodies.viewer().name("indices"),
                    qs            = info.qs().viewer().name("qs"),
                    q_prevs       = info.q_prevs().viewer().name("q_prevs"),
                    aim_transforms = aim_transforms.viewer().name("aim_transforms"),
                    strength_ratios = strength_ratios.viewer().name("strength_ratios"),
                    body_masses = info.body_masses().viewer().name("body_masses"),
                    energies = info.energies().viewer().name("energies"),
                    is_fixed = info.is_fixed().viewer().name("is_fixed")] __device__(int I)
                   {
                       auto  i = indices(I);
                       auto& E = energies(I);

                       if(is_fixed(i))
                       {
                           E = 0.0;
                       }
                       else
                       {
                           Vector12 q      = qs(i);
                           Vector12 q_prev = q_prevs(i);
                           Vector12 q_aim = lerp(q_prev, aim_transforms(I), substep_ratio);
                           Vector12 dq = q - q_aim;
                           Vector2  s  = strength_ratios(I);

                           Float translation_strength = s(0);
                           Float rotation_strength    = s(1);

                           Matrix12x12 M = body_masses(i).to_mat();

                           M.block<3, 3>(0, 0) *= translation_strength;
                           M.block<9, 9>(3, 3) *= rotation_strength;

                           E = 0.5 * dq.transpose() * M * dq;
                       }
                   });
    }

    void do_compute_gradient_hessian(AffineBodyAnimator::GradientHessianInfo& info) override
    {
        using namespace muda;

        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(constrained_bodies.size(),
                   [substep_ratio = info.substep_ratio(),
                    indices       = constrained_bodies.viewer().name("indices"),
                    qs            = info.qs().viewer().name("qs"),
                    q_prevs       = info.q_prevs().viewer().name("q_prevs"),
                    aim_transforms = aim_transforms.viewer().name("aim_transforms"),
                    strength_ratios = strength_ratios.viewer().name("strength_ratios"),
                    body_masses = info.body_masses().viewer().name("body_masses"),
                    gradients = info.gradients().viewer().name("gradients"),
                    hessians  = info.hessians().viewer().name("hessians"),
                    is_fixed = info.is_fixed().viewer().name("is_fixed")] __device__(int I) mutable
                   {
                       auto i = indices(I);

                       Vector12    G;
                       Matrix12x12 H;

                       if(is_fixed(i))
                       {
                           G.setZero();
                           H.setZero();
                       }
                       else
                       {
                           Vector12 q      = qs(i);
                           Vector12 q_prev = q_prevs(i);
                           Vector12 q_aim = lerp(q_prev, aim_transforms(I), substep_ratio);
                           Vector12 dq = q - q_aim;
                           Vector2  s  = strength_ratios(I);

                           Float translation_strength = s(0);
                           Float rotation_strength    = s(1);

                           Matrix12x12 M = body_masses(i).to_mat();

                           M.block<3, 3>(0, 0) *= translation_strength;
                           M.block<9, 9>(3, 3) *= rotation_strength;

                           G = M * dq;
                           H = M;
                       }

                       gradients(I).write(i, G);
                       hessians(I).write(i, i, H);
                   });
    }
};

REGISTER_SIM_SYSTEM(SoftTransformConstraint);
}  // namespace uipc::backend::cuda